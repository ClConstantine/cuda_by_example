#include "hip/hip_runtime.h"
/*
    CUDA流表示一个GPU操作队列,操作以指定顺序执行
*/

#include "../common/book.h"

const int N = 1024 * 1024;
const int FULL_DATA_SIZE = N * 200;

__global__ void kernel(int *a, int *b, int *c){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N){
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;

        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;

        c[idx] = (as + bs) / 2;
    }
}

int main(){

    hipDeviceProp_t prop;
    int whichDevice;
    HANDLE_ERROR(hipGetDevice(&whichDevice));
    HANDLE_ERROR(hipGetDeviceProperties(&prop,whichDevice));
    if(!prop.deviceOverlap){
        printf("Device will not handle overlaps, so no speed up from streams\n");
        return 0;
    }
    /*
        我们需要选择一个支持overlap的设备
        它在执行一个核函数的同时，可以复制数据到设备
        我们使用多个流来实现计算和数据复制的overlap
    */

    hipEvent_t start,stop;
    float elapsedTime;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    
    //初始化流
    hipStream_t stream0, stream1;
    HANDLE_ERROR(hipStreamCreate(&stream0));
    HANDLE_ERROR(hipStreamCreate(&stream1));
    
    int *host_a,*host_b,*host_c;
    int *dev_a0,*dev_b0,*dev_c0;
    int *dev_a1,*dev_b1,*dev_c1;

    HANDLE_ERROR(hipMalloc((void**)&dev_a0,N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b0,N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c0,N * sizeof(int)));

    HANDLE_ERROR(hipMalloc((void**)&dev_a1,N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b1,N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c1,N * sizeof(int)));

    //注意这里分配的内存是页锁定内存
    HANDLE_ERROR(hipHostAlloc((void**)&host_a,FULL_DATA_SIZE * sizeof(int),hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&host_b,FULL_DATA_SIZE * sizeof(int),hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&host_c,FULL_DATA_SIZE * sizeof(int),hipHostMallocDefault));

    for(int i = 0;i < FULL_DATA_SIZE;i++){
        host_a[i] = rand();
        host_b[i] = rand();
    }

    /*
        这里我们不选择将全部数据复制到设备，
        而是选择每次复制N个数据进行分块计算.
        
        hipMemcpyAsync()将以同步的方式复制数据到设备
        任何传递给hipMemcpyAsync()的主机指针都必须是页锁定内存

        异步函数的行为是在流中执行放置一个请求
        当函数返回时，请求可能还没有被执行
        存在的保证是复制操作在下一个流中的操作前执行

        注意这里的kernel函数是在流中执行的
        此时的函数调用是异步的

        我们能够保证流的操作是按照传入的顺序执行的
    */
    HANDLE_ERROR(hipEventRecord(start,0));

    /*
        事实上,虽然在我们流中给他添加了依赖性,
        但是运行时会将操作分别放入内核的复制引擎和核函数执行引擎中,
        依赖性会丢失,CUDA需要处理依赖性.

        所以会存在以下内容

        1 copy a0 to device
        2 copy b0 to device
        3 kernel0 
        4 copy c0 to host ----> relay on the completion of kernel0
        5 copy a1 to device
        6 copy b1 to device
        7 kernel1   
        8 copy c1 to host ----> relay on the completion of kernel1 

        所以执行任然是串行的

        为了让流发挥作用,user需要主动干预

        我们更新模型

        1 copy a0 to device
        2 copy b0 to device
        3 copy a1 to device         kernel0
        4 copy b1 to device         
        5 copy device to c0         kernel1               
        6 copy device to c1

        书本给的模型如下

        1 copy a0 to device
        2 copy a1 to device         
        3 copy b0 to device         5 kernel0    
        4 copy b1 to device         
        7 copy device to c0         6 kernel1               
        8 copy device to c1


    */

    for(int i = 0;i < FULL_DATA_SIZE;i += 2 * N){
        HANDLE_ERROR(hipMemcpyAsync(dev_a0,host_a + i,N * sizeof(int),hipMemcpyHostToDevice,stream0));
        HANDLE_ERROR(hipMemcpyAsync(dev_b0,host_b + i,N * sizeof(int),hipMemcpyHostToDevice,stream0));

        HANDLE_ERROR(hipMemcpyAsync(dev_a1,host_a + i + N,N * sizeof(int),hipMemcpyHostToDevice,stream1));
        HANDLE_ERROR(hipMemcpyAsync(dev_b1,host_b + i + N,N * sizeof(int),hipMemcpyHostToDevice,stream1));

        kernel<<<N/256,256,0,stream0>>>(dev_a0,dev_b0,dev_c0);
        kernel<<<N/256,256,0,stream1>>>(dev_a1,dev_b1,dev_c1);

        HANDLE_ERROR(hipMemcpyAsync(host_c + i,dev_c0,N * sizeof(int),hipMemcpyDeviceToHost,stream0));
        HANDLE_ERROR(hipMemcpyAsync(host_c + i + N,dev_c1,N * sizeof(int),hipMemcpyDeviceToHost,stream1));

    }

    /*
        当for循环结束时，我们需要等待所有的流操作完成
        调用hipStreamSynchronize()来等待流中的所有操作完成
    */
    HANDLE_ERROR(hipStreamSynchronize(stream0));
    HANDLE_ERROR(hipStreamSynchronize(stream1));

    HANDLE_ERROR(hipEventRecord(stop,0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,start,stop));

    printf("Time taken: %3.1f ms\n",elapsedTime);
    //Time taken: 208.6 ms

    HANDLE_ERROR(hipHostFree(host_a));
    HANDLE_ERROR(hipHostFree(host_b));
    HANDLE_ERROR(hipHostFree(host_c));
    
    HANDLE_ERROR(hipFree(dev_a0));
    HANDLE_ERROR(hipFree(dev_b0));
    HANDLE_ERROR(hipFree(dev_c0));
    HANDLE_ERROR(hipFree(dev_a1));
    HANDLE_ERROR(hipFree(dev_b1));
    HANDLE_ERROR(hipFree(dev_c1));


    HANDLE_ERROR(hipStreamDestroy(stream0));
    HANDLE_ERROR(hipStreamDestroy(stream1));

    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    return 0;
}
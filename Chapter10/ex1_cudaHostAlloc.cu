/*
    malloc()和hipHostAlloc()的区别
    malloc()分配的内存是可分页的，
    hipHostAlloc()分配的页锁定内存

    页锁定内存不回将内存页交换到磁盘上，
    操作系统可以安全的访问物理地址

    由于知道物理地址，那么可以使用DMA复制数据
    从主机到设备，而不需要CPU参与，所以将内存固定很重要

    事实上CUDA对于可分页数据的处理仍然是通过DMA的方式
    所以会将数据复制到一块临时锁定内存，再复制到GPU

    但是不能将每一个malloc()替换为hipHostAlloc()
    因为hipHostAlloc()分配的内存是固定的，会很快消耗完memory

    建议仅对hipMemcpy()的源和目的地址使用hipHostAlloc()
    在不使用时立刻free.
*/

#include "../common/book.h"

const int SIZE = 1024 * 1024 * 10;

float cuda_malloc_test(int size,bool up){
    hipEvent_t start,stop;
    int *a,*dev_a;
    float elapsedTime;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    a = (int*)malloc(size*sizeof(int));
    HANDLE_NULL(a);
    HANDLE_ERROR(hipMalloc((void**)&dev_a,size*sizeof(int)));

    HANDLE_ERROR(hipEventRecord(start,0));

    for(int i=0;i<100;i++){
        if(up){
            HANDLE_ERROR(hipMemcpy(dev_a,a,size*sizeof(int),hipMemcpyHostToDevice));
        }else{
            HANDLE_ERROR(hipMemcpy(a,dev_a,size*sizeof(int),hipMemcpyDeviceToHost));
        }
    }

    HANDLE_ERROR(hipEventRecord(stop,0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,start,stop));

    free(a);
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    return elapsedTime;
}

float cuda_host_alloc_test(int size,bool up){
    hipEvent_t start,stop;
    int *a,*dev_a;
    float elapsedTime;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    HANDLE_ERROR(hipHostAlloc((void**)&a,size*sizeof(int),hipHostMallocDefault));
    /*
        分配主机锁定内存
    */
    HANDLE_ERROR(hipMalloc((void**)&dev_a,size*sizeof(int)));

    HANDLE_ERROR(hipEventRecord(start,0));

    for(int i=0;i<100;i++){
        if(up){
            HANDLE_ERROR(hipMemcpy(dev_a,a,size*sizeof(int),hipMemcpyHostToDevice));
        }else{
            HANDLE_ERROR(hipMemcpy(a,dev_a,size*sizeof(int),hipMemcpyDeviceToHost));
        }
    }

    HANDLE_ERROR(hipEventRecord(stop,0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,start,stop));

    HANDLE_ERROR(hipHostFree(a));
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    return elapsedTime;
}



int main(){

    float elapsedTime;
    elapsedTime = cuda_malloc_test(SIZE,true);
    printf("Time using hipMalloc: %3.1f ms\n",elapsedTime);

    elapsedTime = cuda_malloc_test(SIZE,false);
    printf("Time using hipMalloc: %3.1f ms\n",elapsedTime);

    elapsedTime = cuda_host_alloc_test(SIZE,true);
    printf("Time using hipHostAlloc: %3.1f ms\n",elapsedTime);

    elapsedTime = cuda_host_alloc_test(SIZE,false);
    printf("Time using hipHostAlloc: %3.1f ms\n",elapsedTime);

    /*
        10MB
        Time using hipMalloc: 398.3 ms
        Time using hipMalloc: 447.9 ms
        Time using hipHostAlloc: 354.6 ms
        Time using hipHostAlloc: 390.4 ms

        100MB
        Time using hipMalloc: 4062.6 ms
        Time using hipMalloc: 4659.0 ms
        Time using hipHostAlloc: 3442.6 ms
        Time using hipHostAlloc: 3298.2 ms
    */




    return 0;
}

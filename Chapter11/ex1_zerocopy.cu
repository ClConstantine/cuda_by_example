#include "hip/hip_runtime.h"
/*
	零拷贝内存
	在hipHostAlloc中，我们可以通过hipHostMallocMapped来分配内存
	device可以直接访问这块内存，而不需要通过hipMemcpy
	故称为零拷贝内存
	
    但是零拷贝内存会占用系统的物理内存，最终会降低系统性能
    当输入内存和输出内存都只使用一次时，使用零拷贝内存会提高性能
    但是如果输入内存和输出内存都会被多次使用，对于PCIE总线上的优化不如拷贝，最终使得性能下降
*/

#include "../common/book.h"
#include <algorithm> 
#include <cstdlib>
#include <iostream>

const int N = 1024 * 1024 * 50;
const int ROUND = 10;
const int threadsPerBlock = 256;
const int blocksPerGrid = std::min(32,(N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(int size, float *a, float *b, float *c){
    __shared__ float cache[threadsPerBlock];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while(tid < size){
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;

    __syncthreads();

    int i = blockDim.x / 2;
    while(i != 0){
        if(cacheIndex < i){
            cache[cacheIndex] += cache[cacheIndex + i];
            
        }
        __syncthreads();
        i /= 2;
    }
    if(cacheIndex == 0){
        c[blockIdx.x] = cache[0];
    }
}

float malloc_test(int size){
    hipEvent_t start,stop;
    float elapsedTime;
    float *a,*b,*partial_c;
    float *dev_a,*dev_b,*dev_partial_c;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    a = (float*)malloc(size * sizeof(float));
    b = (float*)malloc(size * sizeof(float));
    partial_c = (float*)malloc(blocksPerGrid * sizeof(float));

    HANDLE_ERROR(hipMalloc((void**)&dev_a,size * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b,size * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&dev_partial_c,blocksPerGrid * sizeof(float)));

    for(int i = 0; i < size; i++){
        a[i] = i;
        b[i] = i * 2;
    }

    HANDLE_ERROR(hipEventRecord(start,0));

    HANDLE_ERROR(hipMemcpy(dev_a,a,size * sizeof(float),hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b,b,size * sizeof(float),hipMemcpyHostToDevice));

    dot<<<blocksPerGrid,threadsPerBlock>>>(size,dev_a,dev_b,dev_partial_c);
    //hipMemcpy()会隐式同步
    HANDLE_ERROR(hipMemcpy(partial_c,dev_partial_c,blocksPerGrid * sizeof(float),hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipEventRecord(stop,0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,start,stop));

    float c = 0;
    for(int i = 0; i < blocksPerGrid; i++){
        c += partial_c[i];
    }
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_partial_c));
    
    free(a);
    free(b);
    free(partial_c);

    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    // printf("Value calculated: %f\n",c);

    return elapsedTime;
}

float cuda_host_alloc_test(int size){
    hipEvent_t start,stop;
    float elapsedTime;
    float *a,*b,*partial_c;
    float *dev_a,*dev_b,*dev_partial_c;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    //hipHostMallocMapped标志的内存可以让gpu直接访问    
    HANDLE_ERROR(hipHostAlloc((void**)&a,size * sizeof(float),hipHostMallocMapped | hipHostMallocWriteCombined));
    HANDLE_ERROR(hipHostAlloc((void**)&b,size * sizeof(float),hipHostMallocMapped | hipHostMallocWriteCombined));
    HANDLE_ERROR(hipHostAlloc((void**)&partial_c,blocksPerGrid * sizeof(float),hipHostMallocMapped | hipHostMallocWriteCombined));

    for(int i = 0;i < size;i++){
        a[i] = i;
        b[i] = i * 2;
    }
    /*
        由于gpu的虚拟内存空间和cpu的虚拟内存空间是不同的
        所以需要通过hipHostGetDevicePointer来获取gpu的指针
    */
    HANDLE_ERROR(hipHostGetDevicePointer(&dev_a,a,0));
    HANDLE_ERROR(hipHostGetDevicePointer(&dev_b,b,0));
    HANDLE_ERROR(hipHostGetDevicePointer(&dev_partial_c,partial_c,0));

    HANDLE_ERROR(hipEventRecord(start,0));
    dot<<<blocksPerGrid,threadsPerBlock>>>(size,dev_a,dev_b,dev_partial_c);
    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipEventRecord(stop,0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,start,stop));

    float c = 0;
    for(int i = 0; i < blocksPerGrid; i++){
        c += partial_c[i];
    }

    HANDLE_ERROR(hipHostFree(a));
    HANDLE_ERROR(hipHostFree(b));
    HANDLE_ERROR(hipHostFree(partial_c));

    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    // printf("Value calculated: %f\n",c);
    return elapsedTime;
}

int main(){
   
    hipDeviceProp_t prop;
    int whichDevice;
    HANDLE_ERROR(hipGetDevice(&whichDevice));
    HANDLE_ERROR(hipGetDeviceProperties(&prop,whichDevice));
    if(prop.canMapHostMemory != 1){
        printf("Device cannot map memory.\n");
        return 0;
    }

    float elapsedTime;

    elapsedTime = 0;
    for(int i = 0; i < ROUND; i++){
        elapsedTime += malloc_test(N);
    }
    printf("Time using malloc: %3.1f ms\n",elapsedTime / ROUND);
    // Time using malloc: 45.7 ms
    
    elapsedTime = 0;
    for(int i = 0; i < ROUND; i++){
        elapsedTime += cuda_host_alloc_test(N);
    }
    printf("Time using hipHostAlloc: %3.1f ms\n",elapsedTime / ROUND);
    // Time using hipHostAlloc: 34.9 ms

    return 0;
}
